#include "hip/hip_runtime.h"
/*Copyright 2017 Sushil Dubey, Shashi Dugad, TIFR

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/

// Pixel clusteriser algorithm
// Input: moduleStart[],moduleEnd[], 
// xx[], yy[] with applied ADC threshold
// Output: ClusterId[]


// System includes
#include <stdio.h>
#include <malloc.h>
#include <stdlib.h>
#include <assert.h>
#include <iomanip>
#include <string>
#include <iostream>
#include <fstream>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/unique.h>
#include <thrust/execution_policy.h>
// CUDA runtime
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "CudaError.h"
#include "EventInfoGPU.h"
#include "PixelClusterUtil.h"
#include "PixelClusterMem.h"
#include "PixelCluster.h"

using namespace std;
using namespace thrust;

/*
  The origin of (x,y) was shifted by (1,1) for clustering.
  Reshift the origin to (0,0)
  Input: x[], y[], size
  Output: x[]=x[]-1, y[]=y[]-1; 
*/
__global__ void shift_origin_kernel(uint wordCounter,uint *xx, uint *yy) {
  
  uint gIndex = threadIdx.x + blockIdx.x*blockDim.x;
  if(gIndex<wordCounter) {
    //since bad pixel has x=0,y=0
    if(xx[gIndex]>0) { // either both are 0 or none
      xx[gIndex] = xx[gIndex] -1;
      yy[gIndex] = yy[gIndex] -1;
    }
  }
}

/*
  This kernel sorts the xx[] and yy[] as per the sorted Index[]
  Input: Index[], xx[], yy[]
  Output: xx[], yy[]
*/
__global__ void copy_kernel(const uint *Index,const uint *xx, const uint *yy,
                            const uint *ADC, const uint size, uint *xx1,
                            uint *yy1, uint *ADC1) {
  uint tid = blockDim.x * blockIdx.x + threadIdx.x;
    if(tid < size) {
    xx1[tid] = xx[Index[tid]];
    yy1[tid] = yy[Index[tid]];
    ADC1[tid] = ADC[Index[tid]];
    }
}

/*
  This kernel will check for the subdivision and forms the new cluster
  Input: Index[], xx[], yy[], number of pixels
  Output: gClusterId[]

*/
__global__ void sub_cluster_kernel(const uint *Index, const uint *xx, 
                                     const uint *yy,const uint gpCounter,
                                     uint64 *gClusterId) 
{
  uint tid     = threadIdx.x;
  uint blockid = blockIdx.x;
  uint start   = Index[blockid];
  uint end     = Index[blockid+1];
  __shared__ uint64 old_clusterId;
  __shared__ int nstripx,nstripy;

  // skip the empty clusters
  if(gClusterId[start] == 0) return;
  if(end==(start+1) ) return;

  
  // kernel to handle cluster with size <= 80
  // this is the maximum cluster size
  // found after analysing around 300 events
  if(end-start <= 80) {

    // assuming that cluster size is less than 80
    __shared__ uint xp[162],yp[417];
    __shared__ int tid0x,tid0y;
    __shared__ uint xmin, ymin, xmax, ymax;
    nstripx=0;  nstripy=0;
    tid0x=-1;   tid0y=-1;
    
    #pragma unroll    // tells the compiler to unroll the loop 
    for(uint i=0;i<6;i++) {
      uint gtid = 6*tid + i;  // gtid upto 416 from 80 threads.
      if(gtid<MAX_Y)
        yp[gtid] = 0;
    }
        
    // intialize xp[0] to xp[160] from 80 threads.
    xp[2*tid+ 0]  = 0;
    xp[2*tid + 1] = 0; 
    // find xmin, ymin and subtract from all the pixels
    if(tid==0) {
      xmin = xx[start];
      xmax = xx[start];
      for(int i=1;i<(end-start);i++) {  
        if(xmin > xx[start+i]) {
          xmin = xx[start+i];
        }
        if(xmax<xx[start+i]) {xmax=xx[start+i];}
      }
    }
    if(tid==1) {
      ymin = yy[start];
      for(int i=1;i<(end-start);i++) {  
        if(ymin >yy[start+i]) {
          ymin = yy[start+i];
        }
        if(ymax<yy[start+i]) {ymax=yy[start+i];}
      }
    }
    if(tid==2) {
      xp[160] =0;
      xp[161] =0;
      yp[416] =0;
      old_clusterId = gClusterId[start];
    }
    __syncthreads();  
  
    // to find the projection
    if((start+tid) < end)  {
      uint xc = xx[start + tid] - (xmin-1);
      uint yc = yy[start + tid] - (ymin-1);
      xp[xc] = xc;
      yp[yc] = yc;                                    
    }
    __syncthreads();
  
    if(tid && (nstripx==0 || nstripy==0)){
      if(xp[tid] && !xp[tid - 1]){
        nstripx=1; tid0x=tid;
      }
      if(yp[tid] && !yp[tid - 1]){
        nstripy=1; tid0y=tid;
      }  
    }  // end of if(tid)
    __syncthreads();

    if(tid != tid0x) {
      if(xp[tid] && !xp[tid - 1]){
        nstripx=2;
      }    
    }  // end of if(tid)

    if(tid != tid0y) {
      if(yp[tid] && !yp[tid - 1]) {
        nstripy=2;
      }
    }       
    __syncthreads();
  
    if(nstripx==2 || nstripy==2){ // form cluster only if it is divisible
      if(tid) {
        if(xp[tid] && !xp[tid - 1]) {
          uint i = tid;
          while(xp[i]) {
            xp[i] = tid;
            i++;
            if(i == 162) break;
          }
        }
        if(yp[tid] && !yp[tid - 1]) {
          uint i = tid;
          while(yp[i]) {
            yp[i] = tid;
            i++;
            if(i == 416) break;
          }
        }
      } //end of if(tid)

      // if the difference is grater that 80 then they are not 
      // covered by the thread (rarest case)
      if(xmax-xmin>=blockDim.x) { // if the difference is greater than blockDim.x
        uint ext_tid = blockDim.x + tid;
        if(xp[ext_tid] && !xp[ext_tid-1]) {
          uint i = ext_tid;
          while(xp[i]) {
            xp[i] = ext_tid;
            i++;
            if (i==162) break;
          }
        }
      }

      __syncthreads();

      //assign the cluster id to each pixel
      if((start + tid )< end) {
        uint px=0, py=0;
        uint64 new_clusterId=0;
        px = xp[xx[start + tid]-(xmin-1)] + xmin-1;            
        py = yp[yy[start + tid]-(ymin-1)] + ymin-1;         
        new_clusterId = (old_clusterId & (~uint64(0) << MODULE_shift)) | (py << YCOR_shift) | (px << XCOR_shift);
        if(old_clusterId!=new_clusterId && (px!=0 && py!=0))
          gClusterId[start+tid] = new_clusterId;  
      }
    }//end of if(nstripx==2 || nstripy==2) 
  } // end of if(end-start<=80)
  else if(blockid!=gridDim.x-1){  // sub-cluster kernel to handle cluster with size > 80
    
    __shared__ uint xp[162],yp[417];
    __shared__ uint cluster_size,itrn;
    nstripx=0;  nstripy=0;
    #pragma unroll    // tells the compiler to unroll the loop 
    for(uint i=0;i<6;i++) {
      uint gtid = 6*tid + i;  // gtid upto 416 from 80 threads.
      if(gtid<MAX_Y)
        yp[gtid] = 0;
    }
        
    // intialize yp[0] to yp[160] from 80 threads.
    xp[2*tid+ 0]  = 0;
    xp[2*tid + 1] = 0;  
   
    if(tid==0) {
      xp[160] = 0;
      cluster_size = end-start;
      itrn = cluster_size/blockDim.x + 1;
      old_clusterId = gClusterId[start];
    }
    __syncthreads();  

    //  to find the projection
    #pragma unroll 
    for(uint i=0;i<itrn;i++) {
      uint gtid  = 80*i + tid;
      if(gtid < cluster_size)  {
        uint xc = xx[start+gtid];
        uint yc = yy[start+gtid];
        xp[xc] = xc;
        yp[yc] = yc;                                                                            
      }
      __syncthreads();
    }

    #pragma unroll
    for(uint j =0;j<6; j++) {  // generates gtid form 0.....416 from 80 threads
      uint gtid = 6*tid + j;
      if(gtid && gtid<MAX_Y) {
        if(yp[gtid] && !yp[gtid - 1]) {
          uint i = gtid;
          uint old_nstripy = atomicAdd(&nstripy,1);  // to store the no of xstrip
          while(yp[i]) {
            yp[i] = gtid;
            i++;
            if(i >= MAX_Y) break;
          }
        }
        if(gtid < MAX_X) {
          if(xp[gtid] && !xp[gtid - 1]) {
            uint k = gtid;
            uint old_nstripx = atomicAdd(&nstripx,1);
            while(xp[k]) {
              xp[k] = gtid;
              k++;
              if(k >= MAX_X ) break;
            }
          }
        }//end of if(gtid<161)
      }
      __syncthreads();
    } // end of for loop  

    // assign the cluster id to each pixel only if it divisible 
    if(nstripx>1 || nstripy>1) {
      for(uint i=0;i<itrn;i++) {
        uint gtid = blockDim.x*i + tid;
        if(gtid < cluster_size)  {
          uint px=0, py=0;
          uint64 new_clusterId=0;
          px = xp[xx[start+gtid]];                    // find location of pixel on x strip
          py = yp[yy[start+gtid]];                    // find location of strip on y strip
          new_clusterId = (old_clusterId & (~uint64(0) << MODULE_shift)) | (py << YCOR_shift) | (px << XCOR_shift);
          if(old_clusterId!=new_clusterId && (px!=0 && py!=0))
            gClusterId[start+gtid] = new_clusterId;    
        }
      }
    }
  } // end of else

} // end of sub_cluster

// fills the Index[] with 0, 1,2,3.. upto the size of event
__global__ void createIndex_kernel(const uint wordCounter, uint *Index) {
  uint tid = blockDim.x * blockIdx.x + threadIdx.x;
    if(tid < wordCounter) {
      Index[tid] = tid;
    }
}
void createIndex(const uint wordCounter, uint *Index) {
  // to fill the index array Index[] 
  // which will be used in sort by key 
  int nthreads = 1024;
  int nblocks = wordCounter/nthreads +1;
  createIndex_kernel<<<nblocks, nthreads>>>(wordCounter, Index);
  hipDeviceSynchronize();
  checkCUDAError("Error in createIndex_kernel");
}

/*
  This function sorts the cluster id, removes the duplicate
  and finds the sub-cluster within each cluster
  Input: d_Index[], d_gClusterId[], d_xx[], d_yy[], wordCounter, h_Index[]
  Output:d_xx1[], d_yy1[]
*/
void sub_cluster(uint *d_xx, uint *d_yy,const uint *d_ADC, uint *d_Index, uint64 *d_gClusterId, 
                  const uint wordCounter, uint64 *d_gClusterId1, uint *d_xx1, uint *d_yy1, uint *d_ADC1 ) 
{
  
  //cout<<"Inside sub_cluster function: wordCounter"<<endl;
  createIndex(wordCounter, d_Index);
  
  // get device_ptr needed for thrust operations
  thrust::device_ptr<uint> Index(d_Index); //Index is the index array
  thrust::device_ptr<uint64> gClusterId(d_gClusterId);

  // sort the cluster id by key
  thrust::sort_by_key(gClusterId , gClusterId + wordCounter, Index);
  
  hipMemcpy(d_gClusterId1, d_gClusterId, wordCounter*sizeof(uint64),  hipMemcpyDeviceToDevice );

  // launch kernel for sorting xx[] and yy[]
  uint N_threads = 1024;
  uint N_blocks  = wordCounter / N_threads +1;
  copy_kernel<<<N_blocks,N_threads>>>(d_Index, d_xx, d_yy,d_ADC, wordCounter,d_xx1,d_yy1,d_ADC1); 
  hipDeviceSynchronize();
  checkCUDAError("Error in copy kernel");
  
  // removes the consecutive duplicate
  // new_end.first gives size of gClusterId with no ducplicate
  uint total_cluster=0;
  thrust::pair<thrust::device_ptr<uint64>, thrust::device_ptr<uint> > new_end;
  
  // Fill the index again which will be used for uniuqe 
  createIndex(wordCounter, d_Index);
  
  new_end = thrust::unique_by_key(gClusterId , gClusterId + wordCounter, Index );
  total_cluster = new_end.first - gClusterId;
  checkCUDAError(" Failed after unique operation");
 
  // launch the kernel for subdivision
  dim3 no_threads =  80; // maximum size of cluster found after analysis
  dim3 no_blocks  =  total_cluster;
  // cout<<"Total_clusters: "<<total_cluster<<endl;
  // Ignore first few cluster as they might contain 0s
  sub_cluster_kernel<<<no_blocks,no_threads>>>(d_Index, d_xx1, d_yy1,wordCounter, d_gClusterId1);
  hipDeviceSynchronize();
  checkCUDAError(" Failed after sub-cluster-kernel");
} // End of sort_cluster

__device__ uint64 getClusterId(uint event, uint module, uint y, uint x) {
  uint64 clusterId = ((uint64)event << EVENT_shift) | (module << MODULE_shift) | (y << YCOR_shift) | (x << XCOR_shift);
  return clusterId;
}

/* 
  This is the main kernel for clustarisation
  Inputs:  xx[],yy[],module[], Imodule[]
  Outputs: gClusterId[]
*/
__global__ void cluster_kernel(uint *xx, uint *yy, const int *mIndexStart,
                               const int *mIndexEnd, uint64 *gClusterId) 
{
  __shared__ uint xp[MAX_X+1], yp[MAX_Y+1];   // Array to store x and y projection
  uint moduleId  = blockIdx.x;                // to get block id
  uint event = blockIdx.y;
  uint tid = threadIdx.x;                     // to get thread id
  int  moduleBegin, moduleEnd;
  
  moduleBegin = mIndexStart[event*NMODULE + moduleId];
  moduleEnd   = mIndexEnd[event*NMODULE+ moduleId];

  if(moduleBegin==-1 && moduleEnd==-1) {
    return;
  }
  if(moduleBegin==-1) {
    moduleBegin = moduleEnd;
  }
  if(moduleEnd==-1) {
    moduleEnd = moduleBegin;
  }
  __syncthreads();
  //module contains only one pixel
  if(moduleBegin==moduleEnd) {
    int px = xx[moduleBegin];
    int py = yy[moduleBegin];
    gClusterId[moduleBegin]= getClusterId(event, moduleId, py,px);
    return;
  }
  
  
  uint module_size = (moduleEnd - moduleBegin) + 1;

  yp[tid] = 0;                    // Initialize Y projection to false
  if(tid < MAX_X+1) {xp[tid] = 0;} // Initialize X projection to false
  
  __syncthreads();            // let all threads finish intialization
  // To get projection on x and y axis
  // For loop is used to deal with module with hits more than blockDim.x
  uint noItr = module_size/blockDim.x+1;
  for(uint i=0; i<noItr; i++) {
    uint idx = moduleBegin + tid + i*blockDim.x;
    if(idx <= moduleEnd) {
      uint tx = xx[idx];
      uint ty = yy[idx];
      xp[tx] = tx;
      yp[ty] = ty;
      gClusterId[idx] = 0;
    } // if(tid < module_size)
  } // End of for(uint i=0; i<noItr...
  __syncthreads();

  //  Store the unique strip# for all element in one strip.
  //  Divide the projections array into multiple strips.
  //  Distribute each strip to one thread.
  //  Each thread will process the strip independently
  //  and will store the start location of strip at all location of strip
  //  This procedure will be repeated for both x and y strip.
  if(tid) {
    if(yp[tid] && !yp[tid - 1]) {
      uint i = tid;
      while(yp[i]) {
        yp[i] = tid;
        i++;
        if(i > MAX_Y) break;
      }
    }

    if(tid < MAX_X) {
      if(xp[tid] && !xp[tid - 1]) {
        uint i = tid;
        while(xp[i]) {
          xp[i] = tid;
          i++;
          if(i > MAX_X) break;
        }
      }
    }//end of if(tid<MAX_X){
  } //end of if(tid)
 __syncthreads();
  //assign the cluster id to each pixel
  for(uint i=0; i<noItr; i++) {
    uint idx = moduleBegin + tid + i*blockDim.x;
    if(idx <= moduleEnd) {
      uint px = xp[xx[idx]];                    // find location of pixel on x strip
      uint py = yp[yy[idx]];                    // find location of strip on y strip
      if(px!=0 && py!=0)
        gClusterId[idx] = getClusterId(event, moduleId, py,px);     // gclusterId(idx) = f(event, module,px, py);
    } // if(idx <= moduleEnd)
  } // End of for(uint i=0; i<noItr...  

} //End of cluster_kernel

// assign the index of last cluster
__global__ void init_kernel(const uint wordCounter, const uint total_cluster, uint *Index) {
  Index[total_cluster] = wordCounter;
}

/************* origin of call to the kernel***************/

void PixelCluster_Wrapper(uint *d_xx, uint *d_yy, uint *d_ADC,const uint wordCounter,
                         const int *mIndexStart,const int *mIndexEnd) 
{
    checkCUDAError("Error in RawToDigi, didn't enter in Cluster");    
    cout<<"Clustering started on GPU!"<<endl;

    // hipEventRecord(start);
    hipMemset(d_gClusterId, 0, wordCounter*sizeof(uint64));
    checkCUDAError("Error in setting memory to 0");

    // launch clustering kernel
    int blockX  = MAX_MODULE_SIZE;   // # of blocks in x direction are modules
    int blockY  = NEVENT;            // # of blocks in Y direction are events 
    dim3 threadsize = NO_THREADS;    // # of threads
    dim3 gridsize(blockX, blockY);
    
    cluster_kernel <<< gridsize, threadsize >>>(d_xx, d_yy, mIndexStart, mIndexEnd, d_gClusterId);
    hipDeviceSynchronize();
    checkCUDAError(" Failed after main kernel call");
    
    sub_cluster(d_xx, d_yy, d_ADC, Index, d_gClusterId, wordCounter, d_gClusterId1, d_xx1, d_yy1, d_ADC1);
    
    // FOR CPE formate the output of cluster before giving to CPE
    // sort the clusterIds and corrseponding attributes
    // to get the start and end index of cluster
    createIndex(wordCounter, Index);
    thrust::device_ptr<uint64> ClusterId_ptr(d_gClusterId1);
    thrust::device_ptr<uint> Index_ptr(Index);
    
    thrust::sort_by_key(ClusterId_ptr , ClusterId_ptr + wordCounter, Index_ptr);
    hipMemcpy(d_gClusterId, d_gClusterId1, wordCounter * sizeof(uint64),  hipMemcpyDeviceToDevice );
    // now sort the xx yy and ADC
    uint N_threads = 1024;
    uint N_blocks = wordCounter/N_threads +1;
    copy_kernel<<<N_blocks,N_threads>>>(Index, d_xx1, d_yy1,d_ADC1, wordCounter,d_xx,d_yy,d_ADC); 
    hipDeviceSynchronize();
    checkCUDAError("Error in sorting ");

    uint total_cluster=0;
    thrust::pair<thrust::device_ptr<uint64>, thrust::device_ptr<uint> > new_end;
  
    // Fill the index again which will be used for uniuqe 
    createIndex(wordCounter, Index);
    new_end = thrust::unique_by_key(ClusterId_ptr , ClusterId_ptr + wordCounter, Index_ptr );
    total_cluster = new_end.first - ClusterId_ptr;
    checkCUDAError(" Failed at Clustering");
    
    // Call CPE function
    init_kernel<<<1,1>>>(wordCounter, total_cluster, Index);
    // Index[total_cluster] = wordCounter;
    // since origin is shifted by (1,1) move it back to (0,0) before giving it CPE
    shift_origin_kernel<<<N_blocks, N_threads>>>(wordCounter,d_xx,d_yy); 
    hipDeviceSynchronize();

    // End of Clusterisation and starting CPE
    CPE_wrapper(total_cluster,d_gClusterId1, Index, d_xx, d_yy, d_ADC);
    
} //end of pixel clusterizer

void initDeviceMemCluster() {
    const int MAX_FED  = 150;   // not all are present typically 108
    const int MAX_WORD = 2000;  // don't know the exact max word, for PU70 max was 2900
    const int size = MAX_FED*MAX_WORD*NEVENT*sizeof(uint);
    hipMalloc((void**)&Index , size*sizeof(uint));
    hipMalloc((void**)&d_xx1, size*sizeof(uint));
    hipMalloc((void**)&d_yy1, size*sizeof(uint));
    hipMalloc((void**)&d_gClusterId, size*sizeof(uint64));
    hipMalloc((void**)&d_gClusterId1, size*sizeof(uint64));
    hipMalloc((void**)&d_ADC1,  size*sizeof(uint));
    checkCUDAError("Error in memory allocation for clustering");
}

void freeDeviceMemCluster() {
    hipFree(Index);
    hipFree(d_xx1);
    hipFree(d_yy1);
    hipFree(d_gClusterId);
    hipFree(d_gClusterId1);
    hipFree(d_ADC1);
}