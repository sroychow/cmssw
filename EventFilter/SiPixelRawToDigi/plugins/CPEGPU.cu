#include "hip/hip_runtime.h"
/*Copyright 2017 Sushil Dubey, Shashi Dugad, TIFR

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/

// Cluster paramter Estimation
// Input : ClusterId[], xx[], yy[]
// Output: Hit int global frame

#include <iostream>
#include <fstream>
#include <string>
#include <iomanip>
#include <vector>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "CudaError.h"
#include "PixelClusterUtil.h"
#include "EventInfoGPU.h"
//CPE specific
#include "CPEGPU.h"
// To intitalize memory
#include "CPEGPUMem.h"
//for local to global coordinate conversion
#include "LocalToGlobal.h"
#include "DetParamBits.h"

using namespace std;

__host__ __device__ uint getModule(uint64 clusterId) {
  //uint event = ((clusterId  >> EVENT_shift) & EVENT_mask);
  uint module = ((clusterId >> MODULE_shift) & MODULE_mask);
  //uint xcor  = ((clusterId  >> XCOR_shift) & XCOR_mask);
  //uint ycor  = ((clusterId  >> YCOR_shift) & YCOR_mask);
  return module;
}
__host__ __device__ uint getEvent(uint64 clusterId) {
  uint event = ((clusterId  >> EVENT_shift) & EVENT_mask);
  return event;
}

// CPE kernel for a given cluster, it finds the xhit and yhit
// given in CMSSW CPE package(SiPixelRecHits)
// Input: clusterId, Index, xx, yy, adc
// output: xhit, yhit
__global__ void CPE_kernel(const CPE_cut_Param cpe_cut, const DetDB *detDB, 
                           const uint64 *ClusterId,const uint *Index,const uint *xx,const uint *yy,
                           const uint *adc, const uint numberOfClusters, float *xhit, float *yhit ) 
{
   
  float xmin, xmax;
  float ymin, ymax;
  float Q_l_X, Q_f_X, Q_l_Y, Q_f_Y;
  float sizeX, sizeY;
  LocalPoint lp_min, lp_max;
  LorentzAngle cotAngle;
  float lorentShiftX , lorentShiftY, shiftX, shiftY;
  float theThickness;
  uint startIndex, size, moduleId;

  uint gIndex = threadIdx.x + blockIdx.x*blockDim.x;
  if(gIndex<numberOfClusters) {

    uint64 clusterId = ClusterId[gIndex];
    if(clusterId==0) {
      xhit[gIndex] = 0;
      yhit[gIndex] = 0;
      return ;
    }
  
    startIndex = Index[gIndex];

    size     = Index[gIndex+1] - startIndex;
    moduleId = getModule(clusterId); // to get the moduleId devide by 10^6
    theThickness = (moduleId<1184) ? thicknessBarrel: thicknessForward;
    // Jobs in kernel are 
    // Compute lorentzAngle 

    //for(int i=startIndex; i<startIndex+size; i++) {
      //printf("clusterId: %d  xx: %d   yy:  %d\n",clusterId, xx[i], yy[i] );
    //}
    cotAngle = computeLorentzAngle(detDB,moduleId,startIndex,size, xx, yy,adc); 
    lorentShiftX = detDB->LorentzShiftX[moduleId];  // read from the database
    lorentShiftY = detDB->LorentzShiftY[moduleId];
    
    shiftX = 0.5f*lorentShiftX ;
    shiftY = 0.5f*lorentShiftY;
  
    lorentShiftY = lorentShiftY * widthLAFractionY;
    lorentShiftX = (moduleId<1184) ? lorentShiftX*widthLAFractionX_Barrel: lorentShiftX*widthLAFractionX_Forward;

    // Find xmin, ymin, xmax, ymax 
   
    min_max(startIndex, size, xx, xmin, xmax);
    min_max(startIndex, size, yy, ymin, ymax);
    sizeX = xmax - xmin + 1.0f;
    sizeY = ymax - ymin + 1.0f;
    //printf("xmin: %f, xmax: %f,  ymin: %f, ymax: %f\n",xmin, xmax, ymin, ymax); 

    // Find Q_f and Q_l which depend upon output of step 2

    collectCharge (xx, yy, adc, startIndex, size, xmin, xmax,
                 ymin, ymax, Q_l_X, Q_f_X, Q_l_Y, Q_f_Y );
    //printf("Q_l_X: %f, Q_f_X: %f, Q_l_Y: %f, Q_f_Y: %f",Q_l_X, Q_f_X, Q_l_Y, Q_f_Y);

    // Convert to localPosition in cm depends upon output of steps 2

    lp_min = localPositionInCm( xmin +1.0, ymin+1.0); // use the formula to convert
    lp_max = localPositionInCm( xmax, ymax); // first pix and last pix

    // Compute x_hit using the formula depends upon output of step 1 to 4 ----- by 1st thread
 
    float x_hit=genericPixelHit(sizeX, lp_min.x(), lp_max.x(),
                  Q_f_X, Q_l_X,
                  cotAngle.cotAlpha,
                  pitchX,
                  theThickness,
                  lorentShiftX,
                  isItBigPixelInX((int)xmin),
                  isItBigPixelInX((int)xmax),
                  cpe_cut.the_eff_charge_cut_lowX,
                  cpe_cut.the_eff_charge_cut_highX,
                  cpe_cut.size_cutX
                  );
    x_hit = x_hit + shiftX;
    xhit[gIndex] = x_hit;

  // Compute y_hit using the formula depends upon output of step 1 to 4
    float y_hit=genericPixelHit(sizeY, lp_min.y(), lp_max.y(),
                  Q_f_Y, Q_l_Y,
                  cotAngle.cotBeta,
                  pitchY,
                  theThickness,
                  lorentShiftY,
                  isItBigPixelInY((int)ymin),
                  isItBigPixelInY((int)ymax),
                  cpe_cut.the_eff_charge_cut_lowY,
                  cpe_cut.the_eff_charge_cut_highY,
                  cpe_cut.size_cutY
                  );
    y_hit = y_hit + shiftY;
    yhit[gIndex] = y_hit;
  }
}
// end of CPE_kernel

// device function to calculate the actual pixel hit
// the function is taken from the CMSSW
__device__ float genericPixelHit(uint size, float first_pix, float last_pix,
                      float Q_f, float Q_l, float cot_angle, float pitch,
                      float theThickness, float lorentz_shift,
                      bool first_is_big, bool last_is_big,
                      float eff_charge_cut_low, 
                      float eff_charge_cut_high,
                      float size_cut)

{ 
  float geom_center = 0.5f*(first_pix + last_pix);
  // The case of only one pixel in this projection is separate.  Note that
  // here first_pix == last_pix, so the average of the two is still the
  // center of the pixel.
  if ( size == 1 ) {return geom_center;}

  // Width of the clusters minus the edge (first and last) pixels.
  // In the note, they are denoted x_F and x_L (and y_F and y_L)
  float W_inner = last_pix - first_pix;  // in cm

  // Predicted charge width from geometry
  float W_pred = theThickness * cot_angle - lorentz_shift;// geometric correction (in cm)

  //--- Total length of the two edge pixels (first+last)
  float sum_of_edge = 2.0f;

  if (first_is_big) sum_of_edge += 1.0f;
  if (last_is_big)  sum_of_edge += 1.0f;
  
  //--- The `effective' charge width -- particle's path in first and last pixels only
  if(W_pred<0) W_pred=0-W_pred;
  float W_eff = W_pred - W_inner;

  //--- If the observed charge width is inconsistent with the expectations
  //--- based on the track, do *not* use W_pred-W_innner.  Instead, replace
  //--- it with an *average* effective charge width, which is the average
  //--- length of the edge pixels.
  //
  //  bool usedEdgeAlgo = false;
  if ( (size >= size_cut) || (
       ( W_eff/pitch < eff_charge_cut_low ) |
       ( W_eff/pitch > eff_charge_cut_high ) ) ) {
      W_eff = pitch * 0.5f * sum_of_edge;  // ave. length of edge pixels (first+last) (cm)
  }
    
  //--- Finally, compute the position in this projection
  float Qdiff = Q_l - Q_f;
  float Qsum  = Q_l + Q_f;

  //--- Temporary fix for clusters with both first and last pixel with charge = 0
  if(Qsum==0) Qsum=1.0f;

  float hit_pos = geom_center + 0.5f*(Qdiff/Qsum) * W_eff;
  return hit_pos;
}


// convert local hits to global hits:q

__device__ RecHit toGlobal(const GlobalPosition *gp, const int module,
  const float x, const float y) {
  float xpos = gp[module].xpos;
  float ypos = gp[module].ypos;
  float zpos = gp[module].zpos;
  float r    = gp[module].r;
  Rotation rot = gp[module].Rot;
  float R11  = rot.R11;
  float R12  = rot.R12;
  float R13  = rot.R13;
  float R21  = rot.R21;
  float R22  = rot.R22; 
  float R23  = rot.R23;
  float R31  = rot.R31;
  float R32  = rot.R32;
  float R33  = rot.R33;
  float z =0; // as there is no local z 2D module
  // local to global: Rota[]*local[] + pos[]
  float global_x = (R11*x + R21*y + R31*z) + xpos;
  float global_y = (R12*x + R22*y + R32*z) + ypos;
  float global_z = (R13*x + R23*y + R33*z) + zpos;
  
  RecHit hit;
  hit.x = global_x;
  hit.y = global_y;
  hit.z = global_z;
  // barrel: u=r, v=z, forward the opposite...
  if(module<1184) {
    hit.u = r;
    hit.v = global_z;
    hit.barrel = 1;
  }
  else {
    hit.u = global_z;
    hit.v = r;
  }

  hit.phi  = atanf(hit.y/hit.x);
  hit.theta = atanf(sqrt(powf(hit.x,2)+powf(hit.y, 2))/hit.z);
  // if theta is negative add pi in it
  if(hit.theta<0) hit.theta  = 3.14159 + hit.theta;
  // sign of phi determined as follows found after debugging cmssw
  if(hit.x<0) {
    if(hit.y<0) hit.phi=hit.phi-3.14159;
    else hit.phi = hit.phi + 3.14159; 
  }

  return hit;
}

__global__ void localToGlobal_kernel(const int N, const GlobalPosition *globalPosRot,
  const float *lxhit, const float *lyhit, const uint64 *hitId, RecHit *Hit) {
  int gIndex = threadIdx.x + blockIdx.x*blockDim.x;
  if(gIndex<N) {
    int module = getModule(hitId[gIndex]); // correct the first entry clusterId =0 bad hit
    RecHit hit = toGlobal(globalPosRot, module, lxhit[gIndex], lyhit[gIndex]);
    Hit[gIndex].HitId = hitId[gIndex];
    Hit[gIndex].x = hit.x;
    Hit[gIndex].y = hit.y;
    Hit[gIndex].z = hit.z;
    Hit[gIndex].u = hit.u;
    Hit[gIndex].v = hit.v;
    Hit[gIndex].barrel = hit.barrel;
    Hit[gIndex].phi   = hit.phi;
    Hit[gIndex].theta = hit.theta;
    if(module<1184) {
      Hit[gIndex].layer = getLayer(globalPosRot[module].RawId);
      Hit[gIndex].disk = 0;
    }
    else {
      int disk = getDisk(globalPosRot[module].RawId);
      if (hit.z<0) disk = -disk;
      Hit[gIndex].disk = disk;
      Hit[gIndex].layer= 0;
    }
  }
}


void storeOutput(const int N, const float *lxhit, const float *lyhit, const RecHit *Hit_d) {
  cout<<" store localToGlobal output for validation"<<endl;
  float *lxhit_h, *lyhit_h;
  lxhit_h = (float*)malloc(N*sizeof(float));
  lyhit_h = (float*)malloc(N*sizeof(float));
  RecHit *Hit_h = (RecHit*)malloc(N*sizeof(RecHit));
  GlobalPosition *gp = (GlobalPosition*)malloc(1856*sizeof(GlobalPosition));
  hipMemcpy(gp, globalPosRot, 1856*sizeof(GlobalPosition), hipMemcpyDeviceToHost);
  hipMemcpy(Hit_h, Hit_d, N*sizeof(RecHit), hipMemcpyDeviceToHost);
  hipMemcpy(lxhit_h, lxhit, N*sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(lyhit_h, lyhit, N*sizeof(float), hipMemcpyDeviceToHost);
  checkCUDAError("error in memcpy");
  // ofstream ofile("GlobalHit_GPU_CMSSW.txt");
  // ofile<<"   HitId\t\t localx\t  localy\t  globalx\t   globaly \t  globalz"<<endl;
  ofstream ofile("GPU_CMSSW_localhit_globalhit_phi_theta_for_integration.txt");
  ofstream ofile1("GPU_CMSSW_LocalToGlobal_Output_for_integartion.txt");
  ofile<<"Event   mod   RawId  local_x      local_y     global_x    global_y    global_z    phi    theta "<<endl;
  ofile<<std::fixed;
  ofile<<setprecision(6);
  ofile1<<"Event   mod   RawId     barrel    layer/disk   lx      ly     global_x    global_y    global_z    phi    theta "<<endl;
  ofile1<<std::fixed;
  ofile1<<setprecision(6);

  for(int i=0;i<N;i++) {
    int module = getModule(Hit_h[i].HitId);
    if(Hit_h[i].barrel) {
      ofile1<<setw(4)<<getEvent(Hit_h[i].HitId)<<setw(6)<<module<<setw(11)<<gp[module].RawId;
      ofile1<<setw(6)<<Hit_h[i].barrel<<setw(10)<<Hit_h[i].layer<<setw(13)<<lxhit_h[i]
      <<setw(13)<<lyhit_h[i]<<setw(13)<<Hit_h[i].x<<setw(13)<<Hit_h[i].y<<setw(13)
      <<Hit_h[i].z<<setw(13)<<Hit_h[i].phi<<setw(13)<<Hit_h[i].theta<<endl;
    }
    else {
      ofile1<<setw(4)<<getEvent(Hit_h[i].HitId)<<setw(6)<<module<<setw(11)<<gp[module].RawId;
      ofile1<<setw(6)<<Hit_h[i].barrel<<setw(10)<<Hit_h[i].disk<<setw(13)<<lxhit_h[i]
      <<setw(13)<<lyhit_h[i]<<setw(13)<<Hit_h[i].x<<setw(13)<<Hit_h[i].y<<setw(13)
      <<Hit_h[i].z<<setw(13)<<Hit_h[i].phi<<setw(13)<<Hit_h[i].theta<<endl;
    }
    // for debugging
         
    ofile<<setw(4)<<getEvent(Hit_h[i].HitId)<<setw(6)<<module<<setw(11)<<gp[module].RawId;
    ofile<<setw(13)<<lxhit_h[i]<<setw(13)<<lyhit_h[i]<<setw(13)<<Hit_h[i].x<<setw(13)
    <<Hit_h[i].y<<setw(13)<<Hit_h[i].z<<setw(13)<<Hit_h[i].phi
    <<setw(13)<<Hit_h[i].theta<<endl;
     
    // ofile<<setw(12)<<Hit_h[i].HitId<<setw(6)<<module<<setw(6);
    // ofile<<Hit_h[i].disk<<setw(13)<<Hit_h[i].x<<setw(13)<<Hit_h[i].y<<setw(13)<<Hit_h[i].z<<setw(13)<<gp[module].phi
    // <<setw(13)<<Hit_h[i].phi1<<endl;
  }
  ofile.close();
  ofile1.close();
  free(Hit_h);
  free(lxhit_h);
  free(lyhit_h);
}
//localToGlobal()
//Desc: convert local coordinate of a hit into global hit
//input: Total hits, RotationMatrix, clusterId,local xhit, yhit
//Output: Rechit for each local hit

void localToGlobal(const int N, const GlobalPosition *globalPosRot,
  const uint64 *hitId,const float *lxhit, const float *lyhit,  RecHit *Hit) {
  int threads = 512;
  int blocks  = N/threads +1; 
  cout<<"launching localToGlobal frame ocnversion kernel"<<endl;
  /*hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);*/
  localToGlobal_kernel<<<blocks, threads>>>(N, globalPosRot, lxhit, lyhit, hitId, Hit);
  /*hipDeviceSynchronize();
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  cout<<"Time for localToGlobal coordinate conversion\n Total Hits: "<<N<<" Time(us): "<<milliseconds*1000<<endl;
*/
  checkCUDAError("localToGlobal_kernel failed");
  // only for debugging and validation
  storeOutput(N, lxhit, lyhit, Hit);
  
}

void CPE_wrapper(const uint total_cluster, const uint64 *ClusterId, const uint *Index, const uint *xx, const uint *yy,
                 const uint *adc ) 
{  
  CPE_cut_Param cpe_cut;
  uint no_threads = 512;
  uint no_blocks = total_cluster/no_threads +1;
  // xhit_d, yhit_d, contains output
  CPE_kernel<<<no_blocks, no_threads>>>(cpe_cut,detDB,ClusterId, Index, xx, yy, adc,total_cluster, xhit_d, yhit_d);
  hipDeviceSynchronize();
  checkCUDAError("Error in CPE_kernel");
  cout<<"CPE execution finished!\n";
  //convert local hit in global frame
  localToGlobal(total_cluster, globalPosRot, ClusterId, xhit_d, yhit_d, Hit);
  cout<<"All the hits converted into global frame successfully!"<<endl;
}

// compute cot alpha and beta for each cluster
// formula to calculate cot alpha and cot beta is taken from
// http://cmslxr.fnal.gov/source/RecoLocalTracker/SiPixelRecHits/src/PixelCPEBase.cc?v=CMSSW_8_1_0#0325
// https://cmssdt.cern.ch/lxr/source/DataFormats/SiPixelCluster/interface/SiPixelCluster.h#0104
// Input: as shown in fucntion
// Output: LorentzAngle cotAlpha and cotBeta
__device__ LorentzAngle computeLorentzAngle(const DetDB *detDB, const uint moduleId,
                        const uint startIndex, const uint size, const uint *xx,
                        const uint *yy, const uint *adc ) 
{

  float totalCharge = 0.0f;
  float xc=0.0f, yc=0.0f;
  uint i= startIndex;
  uint end = startIndex+size;
  for (; i<end; i++) {
    xc += (xx[i] + 0.5f)*adc[i];
    yc += (yy[i] + 0.5f)*adc[i];
    totalCharge += adc[i];
  }
  xc = xc/totalCharge;
  yc = yc/totalCharge;
  LocalPoint lp = localPositionInCm(xc, yc);

  float gvx = lp.x() - detDB->X0[moduleId];
  float gvy = lp.y() - detDB->Y0[moduleId];
  float gvz = -1.f/detDB->Z0[moduleId];
  float cot_aplha = gvx*gvz;
  float cot_beta  = gvy*gvz;
  LorentzAngle la = { cot_aplha, cot_beta }; 
  return la;
}

// device function to find min max
__device__ void min_max(uint startIndex,uint size,
                        const uint *xx, float &xmin,float &xmax) 
{
  
  xmin = xx[startIndex];
  xmax = xx[startIndex];
  uint i = startIndex+1;
  for(; i<(startIndex+ size); i++) {
    if(xmin>float(xx[i])) xmin = xx[i];
    if(xmax<float(xx[i])) xmax = xx[i];
  }
}

// device function to collect charges on
// the edge of the cluster
__device__ void 
collectCharge (const uint *xx, const uint *yy, const uint *adc,
               uint startIndex, uint size, float xmin, float xmax,
               float ymin, float ymax, float &Q_l_X, float &Q_f_X,
               float &Q_l_Y, float &Q_f_Y ) 
{
  Q_f_X = 0.0f;
  Q_l_X = 0.0f;
  Q_f_Y = 0.0f;
  Q_l_Y = 0.0f;
  float pix_adc = 0.0f;
  uint i=startIndex;
  for(; i<(startIndex+size); i++) {
    // upper cut is put on pixel charge but does not affect the result much 
    pix_adc = adc[i];//pixel.ADC > 13200.0f ? 13200.0f:pixel.ADC;
    if((float(xx[i])==xmin)) Q_f_X+= pix_adc;
    if((float(xx[i])==xmax)) Q_l_X+= pix_adc;
    if((float(yy[i])==ymin)) Q_f_Y+= pix_adc;
    if((float(yy[i])==ymax)) Q_l_Y+= pix_adc;
  }
} 

// this function converts pixel coordinates row and col in cm
// multiply the row and col by pitch size to convert in cm
// Input: x(0-159), y(0-145)
// Output: x(-0.81 cm to +0.81 cm), y(-3.24 cm to +3.24 cm) 
__device__ LocalPoint localPositionInCm(float x, float y) {
  //  m_xoffset = -(m_nrows + BIG_PIX_PER_ROC_X*m_nrows/ROWS_PER_ROC)/2. * 
  //  m_pitchx;
  //  m_yoffset = -(m_ncols + BIG_PIX_PER_ROC_Y*m_ncols/COLS_PER_ROC)/2. * 
  //  m_pitchy;
  // m_nrows = 160, BIG_PIX_PER_ROC_X=1, ROWS_PER_ROC=80,m_pitchx=0.01
  // m_ncols = 416, BIG_PIX_PER_ROC_Y=2, COLS_PER_ROC = 52, m_pitchy = 0.015
  // after calculating  
  float m_xoffset = 0.81f;
  float m_yoffset = 3.24f;
  // As big pixel issue is corrected in CMSSW_9_2_0
  int binoffx = int( x );        // truncate to int
  float fractionX = x - float(binoffx); // find the fraction 
  float local_pitchx = pitchX;   // default pitch
   
  if (binoffx>80) {            // ROC 1 - handles x on edge cluster
    binoffx=binoffx+2;
  } 
  else if (binoffx==80) {    // ROC 1
    binoffx=binoffx+1;
    local_pitchx *= 2;
  }
  else if (binoffx==79) {      // ROC 0
    binoffx=binoffx+0;
    local_pitchx *= 2;    
  } 
  // The final position in local coordinates 
  //float lpX = float( binoffx * m_pitchx ) + fractionX * local_pitchx + m_xoffset;
  float xcm = float(binoffx * pitchX) + fractionX * local_pitchx - m_xoffset;

  int binoffy = int( y );        // truncate to int
  float fractionY = y - float(binoffy); // find the fraction 
  float local_pitchy = pitchY;   // defaultpitch
  // 415 is last big pixel, 416 and above do not exists!
  //constexpr int bigYIndeces[]{0,51,52,103,104,155,156,207,208,259,260,311,312,363,364,415};
  //auto const j = std::lower_bound(std::begin(bigYIndeces),std::end(bigYIndeces),binoffy);
  //if (*j==binoffy) { local_pitchy  *= 2 ;}
  //binoffy += (j-bigYIndeces);
  if(binoffy>416) binoffy=432; //this is due to the bug in cmssw cpe, since origin is shifted by 1 remove this in cmssw
  else if(!(binoffy%52)) {
    binoffy += ((int)(binoffy/52))*2;
    local_pitchy  *= 2 ;
  }
  else {
    binoffy += ((int)(binoffy/52))*2 +1;
    if(!(binoffy+1)%52) local_pitchy  *= 2 ;
  }
  // The final position in local coordinates 
  float ycm = float(binoffy*pitchY) + fractionY*local_pitchy - m_yoffset;

  //float xcm = x*pitchX - m_xoffset;
  //float ycm = y*pitchY - m_yoffset;
  LocalPoint lp;
  lp.xcm=xcm;
  lp.ycm=ycm;
  return lp;
}

//-------------------------------------------------------------
// Return the BIG pixel information for a given pixel
// reference: http://cmslxr.fnal.gov/source/Geometry/TrackerGeometryBuilder/interface/RectangularPixelTopology.h?v=CMSSW_9_2_0#0119
__device__ bool isItBigPixelInX( const int ixbin ) {
  return (( ixbin == 79 ) || ( ixbin == 80 ));
}

__device__ bool isItBigPixelInY( const int iybin ) {
  int iybin0 = iybin%52;
  return(( iybin0 == 0 ) || ( iybin0 == 51 ));
}

void initDeviceMemCPE() {
  const int MAX_CLUSTER = 20000*NEVENT;
  hipMalloc((void**)&xhit_d, MAX_CLUSTER*sizeof(float));
  hipMalloc((void**)&yhit_d, MAX_CLUSTER*sizeof(float));
  hipMallocManaged((void**)&detDB, sizeof(DetDB));
  uploadCPE_db(detDB);
  // allocate memory to hold the global hits and other parameter 
  const int size = MAX_CLUSTER*sizeof(RecHit);
  hipMalloc((void**)&Hit, size);
  hipMalloc((void**)&globalPosRot, NMODULE*sizeof(GlobalPosition));
  // upload global position and rotation matrix for each module
  uploadGlobal_Positon_Rotation_Matrix(globalPosRot);
}
void freeDeviceMemCPE() {
  hipFree(xhit_d);
  hipFree(yhit_d);
  hipFree(detDB);
  hipFree(Hit);
  hipFree(globalPosRot);
}

// upload the CPE database to the GPU memory
// they are constant for the module
void uploadCPE_db(DetDB *detDB) {
  uint moduleId,rawId,i=0;
  float X0, Y0, Z0, Rdet, Zdet, LShiftX, LShiftY;
  ifstream ifile("Pixel_CPE_Phase1_database_C_920.dat");
  if(!ifile) {
    cout<<"CPE database file does not exit !"<<endl;
    cout<<"File: Pixel_CPE_Phase1_database_C_920.dat"<<endl;
    exit(1);
  }
  string str;
  getline(ifile, str);
  while(!ifile.eof()) {
    ifile>>moduleId>>rawId>>X0>>Y0>>Z0>>Rdet>>Zdet>>LShiftX>>LShiftY;
    detDB->RawId[i] = rawId;
    detDB->X0[i] = X0;
    detDB->Y0[i] = Y0;
    detDB->Z0[i] = Z0;
    detDB->LorentzShiftX[i] = LShiftX;
    detDB->LorentzShiftY[i] = LShiftY;
    i++ ;
  }
  ifile.close();
  cout<<"CPE database uploaded successfully ! "<<endl;
}

// upload the global position and rotation matrix for 
// local to global coordinate coversion
void uploadGlobal_Positon_Rotation_Matrix(GlobalPosition *globalPosRot) {
  GlobalPosition *gp;
  gp = (GlobalPosition*)malloc(NMODULE*sizeof(GlobalPosition));
  // read the file and upload
  ifstream ifile("Global_Position_Rotation_forL2G.dat");
  if(!ifile) {
    cout<<"File not found: Global_Position_Rotation_forL2G.dat"<<endl;
    cout<<"exit...";
    exit(-1);
  }
  string line;
  getline(ifile, line);
  for(int i=0;i<NMODULE;i++) {
    ifile>>gp[i].RawId>>gp[i].xpos>>gp[i].ypos>>gp[i].zpos>>gp[i].r>>gp[i].phi;
    ifile>>gp[i].Rot.R11>>gp[i].Rot.R12>>gp[i].Rot.R13;
    ifile>>gp[i].Rot.R21>>gp[i].Rot.R22>>gp[i].Rot.R23;
    ifile>>gp[i].Rot.R31>>gp[i].Rot.R32>>gp[i].Rot.R33;
  }
  hipMemcpy(globalPosRot, gp, NMODULE*sizeof(GlobalPosition), hipMemcpyHostToDevice);
  free(gp);
  cout<<"Global position and Rotation matrix uploaded successfully!"<<endl;
}
